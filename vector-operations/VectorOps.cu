#include "hip/hip_runtime.h"
//
// Created by leo on 2/17/24.
//

#include <iostream>
#include <algorithm>
#include <cstdlib>
#include "VectorOps.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "boost/program_options/variables_map.hpp"
#include "boost/program_options/options_description.hpp"

namespace po = boost::program_options;

__global__ void add_vecs(float *vec1, float *vec2, float *outVec) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    outVec[i] = vec1[i] + vec2[i];
}

__global__ void sub_vecs(float *vec1, float *vec2, float *outVec) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    outVec[i] = vec1[i] - vec2[i];
}

__global__ void dot_product_vecs(float *vec1, float *vec2, float *outVec) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    outVec[i] = vec1[i] * vec2[i];
}

void print_vector(float* vec, int size) {
    std::cout << "[ ";
    if (size <= 15) {
        for(int i=0;i<size;++i)
            std::cout << vec[i] << ' ';
    } else {
        for(int i=0;i<15;++i)
            std::cout << vec[i] << ' ';
        std::cout << "... " << vec[size-1];
    }
    std::cout << " ]" << std::endl;
}

int VectorOps::main(const po::variables_map &vm) {

    uint amountElements = 15;
    if (vm.count("numElements")) {
        amountElements = vm["numElements"].as<int>();
        std::cout << "Randomizing " << amountElements << " elements and performing operations" << std::endl;
    }

    float *vec1;
    float *vec2;
    hipMalloc(&vec1, amountElements * sizeof(float));
    hipMalloc(&vec2, amountElements * sizeof(float));
    float *add_result;
    float *sub_result;
    float *dot_prod_result;
    hipMalloc(&add_result, amountElements * sizeof(float));
    hipMalloc(&sub_result, amountElements * sizeof(float));
    hipMalloc(&dot_prod_result, amountElements * sizeof(float));

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    hiprandGenerateUniform(gen, vec1, amountElements);
    hiprandGenerateUniform(gen, vec2, amountElements);

    hiprandDestroyGenerator(gen);

    int threadsPerBlock = 256;
    int blocksPerGrid = (amountElements + threadsPerBlock - 1) / threadsPerBlock;

    float* host_vec1 = new float[amountElements];
    hipMemcpy(host_vec1, vec1, amountElements * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Vec1: ";
    print_vector(host_vec1,amountElements);

    float* host_vec2 = new float[amountElements];
    hipMemcpy(host_vec2, vec2, amountElements * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Vec2: ";
    print_vector(host_vec2,amountElements);

    delete[] host_vec1;
    delete[] host_vec2;

    // Addition
    add_vecs<<<blocksPerGrid, threadsPerBlock>>>(vec1, vec2, add_result);

    // Subtraction
    sub_vecs<<<blocksPerGrid, threadsPerBlock>>>(vec1, vec2, sub_result);

    // Dot product
    dot_product_vecs<<<blocksPerGrid, threadsPerBlock>>>(vec1, vec2, dot_prod_result);

    // Synchronize Device
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    float* host_add_result = new float[amountElements];
    hipMemcpy(host_add_result, add_result, amountElements * sizeof(float), hipMemcpyDeviceToHost);
    float* host_sub_result = new float[amountElements];
    hipMemcpy(host_sub_result, sub_result, amountElements * sizeof(float), hipMemcpyDeviceToHost);
    float* host_dot_prod_result = new float[amountElements];
    hipMemcpy(host_dot_prod_result, dot_prod_result, amountElements * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Addition output vector: ";
    print_vector(host_add_result, amountElements);

    std::cout << "Subtraction output vector: ";
    print_vector(host_sub_result, amountElements);

    float dotProduct = 0.0f;
    for (int i = 0; i < amountElements; i++) {
        dotProduct += host_dot_prod_result[i];
    }

    printf("Dot product: %.5f\n", dotProduct);

    hipFree(vec1);
    hipFree(vec2);
    hipFree(add_result);
    hipFree(sub_result);
    hipFree(dot_prod_result);

    delete[] host_add_result;
    delete[] host_sub_result;
    delete[] host_dot_prod_result;

    return 0;
}

VectorOps::VectorOps() {

}

void VectorOps::addParams(po::options_description *desc) {
    desc->add_options() ("numElements,n", po::value<int>()->implicit_value(15), "Number of elements to generate");
}